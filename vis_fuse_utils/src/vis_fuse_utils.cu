#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <igl/Hit.h>
#include <igl/embree/EmbreeIntersector.h>
#include <igl/parallel_for.h>
#include <igl/ray_mesh_intersect.h>
#include <pybind11/eigen.h>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <stdio.h>

#define BLOCK_DIM 16

/**
 * Computes the squared Euclidean distance matrix between the query points and
 * the reference points.
 *
 * @param ref          refence points stored in the global memory
 * @param ref_width    number of reference points
 * @param ref_pitch    pitch of the reference points array in number of column
 * @param query        query points stored in the global memory
 * @param query_width  number of query points
 * @param query_pitch  pitch of the query points array in number of columns
 * @param height       dimension of points = height of texture `ref` and of the
 * array `query`
 * @param dist         array containing the query_width x ref_width computed
 * distances
 */
__global__ void ComputeDistances(float *ref, int ref_width, int ref_pitch,
                                 float *query, int query_width, int query_pitch,
                                 int height, float *dist) {
  // Declaration of the shared memory arrays As and Bs used to store the
  // sub-matrix of A and B
  __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
  __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

  // Sub-matrix of A (begin, step, end) and Sub-matrix of B (begin, step)
  __shared__ int begin_A;
  __shared__ int begin_B;
  __shared__ int step_A;
  __shared__ int step_B;
  __shared__ int end_A;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Initializarion of the SSD for the current thread
  float ssd = 0.f;

  // Loop parameters
  begin_A = BLOCK_DIM * blockIdx.y;
  begin_B = BLOCK_DIM * blockIdx.x;
  step_A = BLOCK_DIM * ref_pitch;
  step_B = BLOCK_DIM * query_pitch;
  end_A = begin_A + (height - 1) * ref_pitch;

  // Conditions
  int cond0 = (begin_A + tx < ref_width);  // used to write in shared memory
  int cond1 = (begin_B + tx <
               query_width);  // used to write in shared memory & to
                              // computations and to write in output array
  int cond2 =
      (begin_A + ty <
       ref_width);  // used to computations and to write in output matrix

  // Loop over all the sub-matrices of A and B required to compute the block
  // sub-matrix
  for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {
    // Load the matrices from device memory to shared memory; each thread loads
    // one element of each matrix
    if (a / ref_pitch + ty < height) {
      shared_A[ty][tx] = (cond0) ? ref[a + ref_pitch * ty + tx] : 0;
      shared_B[ty][tx] = (cond1) ? query[b + query_pitch * ty + tx] : 0;
    } else {
      shared_A[ty][tx] = 0;
      shared_B[ty][tx] = 0;
    }

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Compute the difference between the two matrixes; each thread computes one
    // element of the block sub-matrix
    if (cond2 && cond1) {
      for (int k = 0; k < BLOCK_DIM; ++k) {
        float tmp = shared_A[k][ty] - shared_B[k][tx];
        ssd += tmp * tmp;
      }
    }

    // Synchronize to make sure that the preceeding computation is done before
    // loading two new sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory; each thread writes one element
  if (cond2 && cond1) {
    dist[(begin_A + ty) * query_pitch + begin_B + tx] = ssd;
  }
}

/**
 * For each query / reference point (i.e. each row / column) finds the smallest
 * distances of the distance matrix and their respective indexes and gathers
 * them at the top of the 2 arrays.
 *
 * @param dist         distance matrix
 * @param dist_pitch   pitch of the distance matrix given in number of columns
 * @param closest_indices        closest_indices matrix
 * @param closest_dists        closest_dists matrix
 * @param width        width of the distance matrix and of the index matrix
 * @param height       height of the distance matrix
 */
__global__ void ComputeClosestIndex(float *dist, int dist_pitch,
                                    int *closest_indices, float *closest_dists,
                                    int width, int height, bool is_col) {
  // Column position
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  // Do nothing if we are out of bounds
  if (index < (is_col ? width : height)) {
    // Pointer shift
    float *p_dist = dist + (is_col ? index : index * dist_pitch);
    int *closest_index = closest_indices + index;
    float *closest_dist = closest_dists + index;

    // initialize index
    *closest_index = -1;
    *closest_dist = INFINITY;

    // Go through all points
    for (int i = 0; i < (is_col ? height : width); ++i) {
      // Store current distance and associated index
      float curr_dist = sqrt(p_dist[(is_col ? i * dist_pitch : i)]);

      if (curr_dist < *closest_dist) {
        *closest_dist = curr_dist;
        *closest_index = i;
      }
    }
  }
}

// adapted from
// https://github.com/vincentfpgarcia/kNN-CUDA/blob/master/code/knncuda.h#L14
void ClosestNeighbourCuda(const float *ref, int ref_nb, const float *query,
                          int query_nb, int dim, float *knn_dist_ref,
                          int *knn_index_ref, float *knn_dist_query,
                          int *knn_index_query) {
  // Constants
  const unsigned int size_of_float = sizeof(float);
  const unsigned int size_of_int = sizeof(int);

  // Return variables
  hipError_t err0, err1, err2, err3, err4, err5, err6;

  // Check that we have at least one CUDA device
  int nb_devices;
  err0 = hipGetDeviceCount(&nb_devices);
  if (err0 != hipSuccess || nb_devices == 0) {
    throw std::runtime_error("ERROR: No CUDA device found\n");
  }

  // Select the first CUDA device as default
  err0 = hipSetDevice(0);
  if (err0 != hipSuccess) {
    throw std::runtime_error("ERROR: Cannot set the chosen CUDA device\n");
  }

  // Allocate global memory
  float *ref_dev = nullptr;
  float *query_dev = nullptr;
  float *dist_dev = nullptr;
  size_t ref_pitch_in_bytes;
  size_t query_pitch_in_bytes;
  size_t dist_pitch_in_bytes;

  err0 = hipMallocPitch((void **)&ref_dev, &ref_pitch_in_bytes,
                         ref_nb * size_of_float, dim);
  err1 = hipMallocPitch((void **)&query_dev, &query_pitch_in_bytes,
                         query_nb * size_of_float, dim);
  err2 = hipMallocPitch((void **)&dist_dev, &dist_pitch_in_bytes,
                         query_nb * size_of_float, ref_nb);

  int *closest_index_ref_dev = nullptr;
  float *closest_dist_ref_dev = nullptr;
  int *closest_index_query_dev = nullptr;
  float *closest_dist_query_dev = nullptr;

  err3 = hipMalloc((void **)&closest_index_ref_dev, ref_nb * size_of_int);
  err4 = hipMalloc((void **)&closest_dist_ref_dev, ref_nb * size_of_float);
  err5 = hipMalloc((void **)&closest_index_query_dev, query_nb * size_of_int);
  err6 = hipMalloc((void **)&closest_dist_query_dev, query_nb * size_of_float);

  auto free = [&]() {
    hipFree(ref_dev);
    hipFree(query_dev);
    hipFree(dist_dev);
    hipFree(closest_index_ref_dev);
    hipFree(closest_dist_ref_dev);
    hipFree(closest_index_query_dev);
    hipFree(closest_dist_query_dev);
  };

  if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess ||
      err3 != hipSuccess || err4 != hipSuccess || err5 != hipSuccess ||
      err6 != hipSuccess) {
    free();
    throw std::runtime_error("ERROR: Memory allocation error\n");
  }

  // Deduce pitch values
  size_t ref_pitch = ref_pitch_in_bytes / size_of_float;
  size_t query_pitch = query_pitch_in_bytes / size_of_float;
  size_t dist_pitch = dist_pitch_in_bytes / size_of_float;

  // Check pitch values
  if (query_pitch != dist_pitch) {
    free();
    throw std::runtime_error("ERROR: Invalid pitch value\n");
  }

  // Copy reference and query data from the host to the device
  err0 = hipMemcpy2D(ref_dev, ref_pitch_in_bytes, ref, ref_nb * size_of_float,
                      ref_nb * size_of_float, dim, hipMemcpyHostToDevice);
  err1 = hipMemcpy2D(query_dev, query_pitch_in_bytes, query,
                      query_nb * size_of_float, query_nb * size_of_float, dim,
                      hipMemcpyHostToDevice);
  if (err0 != hipSuccess || err1 != hipSuccess) {
    free();
    throw std::runtime_error(
        "ERROR: Unable to copy data from host to device\n");
  }

  // Compute the squared Euclidean distances
  dim3 block0(BLOCK_DIM, BLOCK_DIM, 1);
  dim3 grid0(query_nb / BLOCK_DIM, ref_nb / BLOCK_DIM, 1);
  if (query_nb % BLOCK_DIM != 0) grid0.x += 1;
  if (ref_nb % BLOCK_DIM != 0) grid0.y += 1;
  ComputeDistances<<<grid0, block0>>>(ref_dev, ref_nb, ref_pitch, query_dev,
                                      query_nb, query_pitch, dim, dist_dev);
  if (hipGetLastError() != hipSuccess) {
    free();
    throw std::runtime_error(
        "ERROR: Unable to execute ComputeDistances kernel\n");
  }

  // Retrieve closest dist and index
  dim3 block_ref(256, 1, 1);
  dim3 grid_ref(ref_nb / 256, 1, 1);
  if (ref_nb % 256 != 0) grid_ref.x += 1;
  ComputeClosestIndex<<<grid_ref, block_ref>>>(
      dist_dev, dist_pitch, closest_index_ref_dev, closest_dist_ref_dev,
      query_nb, ref_nb, false);
  if (hipGetLastError() != hipSuccess) {
    free();
    throw std::runtime_error("ERROR: Unable to execute closest_index kernel\n");
  }

  dim3 block_query(256, 1, 1);
  dim3 grid_query(query_nb / 256, 1, 1);
  if (query_nb % 256 != 0) grid_query.x += 1;
  ComputeClosestIndex<<<grid_query, block_query>>>(
      dist_dev, dist_pitch, closest_index_query_dev, closest_dist_query_dev,
      query_nb, ref_nb, true);
  if (hipGetLastError() != hipSuccess) {
    free();
    throw std::runtime_error("ERROR: Unable to execute closest_index kernel\n");
  }

  // Copy the smallest distances / indexes from the device to the host
  err0 = hipMemcpy(knn_dist_ref, closest_dist_ref_dev, ref_nb * size_of_float,
                    hipMemcpyDeviceToHost);
  err1 = hipMemcpy(knn_index_ref, closest_index_ref_dev, ref_nb * size_of_int,
                    hipMemcpyDeviceToHost);
  err2 = hipMemcpy(knn_dist_query, closest_dist_query_dev,
                    query_nb * size_of_float, hipMemcpyDeviceToHost);
  err3 = hipMemcpy(knn_index_query, closest_index_query_dev,
                    query_nb * size_of_int, hipMemcpyDeviceToHost);

  if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess ||
      err3 != hipSuccess) {
    free();
    throw std::runtime_error(
        "ERROR: Unable to copy data from device to host\n");
  }

  // Memory clean-up
  free();
}

namespace py = pybind11;

py::list ComputeClosestNeighbour(py::array_t<float> ref_pts,
                                 py::array_t<float> query_pts) {
  auto r_ref_pts = ref_pts.unchecked<2>();
  auto r_query_pts = query_pts.unchecked<2>();

  if (r_ref_pts.shape(1) != r_query_pts.shape(1)) {
    throw std::runtime_error(
        "Two point sets shoud have same feature dimension");
  }

  int ref_nb = r_ref_pts.shape(0);
  int query_nb = query_pts.shape(0);
  int dim = r_ref_pts.shape(1);

  float *closest_dist_ref = (float *)malloc(ref_nb * sizeof(float));
  int *closest_index_ref = (int *)malloc(ref_nb * sizeof(int));
  float *closest_dist_query = (float *)malloc(query_nb * sizeof(float));
  int *closest_index_query = (int *)malloc(query_nb * sizeof(int));

  ClosestNeighbourCuda(ref_pts.data(), ref_nb, query_pts.data(), query_nb, dim,
                       closest_dist_ref, closest_index_ref, closest_dist_query,
                       closest_index_query);

  py::list results_list;

  auto closest_dist_ref_py = py::array_t<float>(ref_nb, closest_dist_ref);
  auto closest_index_ref_py = py::array_t<int>(ref_nb, closest_index_ref);
  auto closest_dist_query_py = py::array_t<float>(query_nb, closest_dist_query);
  auto closest_index_query_py = py::array_t<int>(query_nb, closest_index_query);

  results_list.append(closest_dist_ref_py);
  results_list.append(closest_index_ref_py);
  results_list.append(closest_dist_query_py);
  results_list.append(closest_index_query_py);

  return results_list;
}

using RowMatrixXd =
    Eigen::Matrix<double, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor>;
using RowMatrixXi =
    Eigen::Matrix<int, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor>;

RowMatrixXi SampleOcclusionEmbree(Eigen::Ref<const RowMatrixXd> V,
                                  Eigen::Ref<const RowMatrixXi> F,
                                  Eigen::Ref<const RowMatrixXd> origins,
                                  Eigen::Ref<const RowMatrixXd> dirs) {
  igl::embree::EmbreeIntersector intersector;
  intersector.init(V.cast<float>(), F.cast<int>(), true);

  RowMatrixXi occlusion(origins.rows(), dirs.rows());
  igl::parallel_for(
      origins.rows() * dirs.rows(),
      [&](int i) {
        int origin_idx = i / dirs.rows();
        int dir_idx = i % dirs.rows();
        igl::Hit hit;
        bool is_hit =
            intersector.intersectRay(origins.row(origin_idx).cast<float>(),
                                     dirs.row(dir_idx).cast<float>(), hit);
        occlusion.coeffRef(origin_idx, dir_idx) = static_cast<int>(is_hit);
      },
      1000);
  return occlusion;
}

RowMatrixXi SampleOcclusion(Eigen::Ref<const RowMatrixXd> V,
                            Eigen::Ref<const RowMatrixXi> F,
                            Eigen::Ref<const RowMatrixXd> origins,
                            Eigen::Ref<const RowMatrixXd> dirs) {
  RowMatrixXi occlusion(origins.rows(), dirs.rows());
  igl::parallel_for(
      origins.rows() * dirs.rows(),
      [&](int i) {
        int origin_idx = i / dirs.rows();
        int dir_idx = i % dirs.rows();
        igl::Hit hit;
        bool is_hit =
            igl::ray_mesh_intersect(origins.row(origin_idx).cast<float>(),
                                    dirs.row(dir_idx).cast<float>(), V, F, hit);
        occlusion.coeffRef(origin_idx, dir_idx) = static_cast<int>(is_hit);
      },
      1000);
  return occlusion;
}

PYBIND11_MODULE(vis_fuse_utils_bind, m) {
  m.def("compute_closest_neighbor", &ComputeClosestNeighbour,
        "Compute mutual closest point distance and index. Modified from "
        "https://github.com/vincentfpgarcia/kNN-CUDA");
  m.def("sample_occlusion", &SampleOcclusion,
        py::return_value_policy::reference_internal, "Sample occlusions");
  m.def("sample_occlusion_embree", &SampleOcclusionEmbree,
        py::return_value_policy::reference_internal,
        "Sample occlusions using embree");
}
